// ------------------------------------------------------------
//  作者：ChatGPT
//  功能：演示如何使用 cuBLAS‑Lt（cublasLtMatmul）完成
//        单精度 (FP32) 与半精度 (FP16) 的矩阵乘法，并与 CPU 结果对比。
//  说明：
//  * 采用列主序（column‑major）布局，符合 cuBLAS 默认约定。
//  * 通过 cublasLtMatrixLayout / cublasLtMatmulDesc 等 API 显式描述矩阵。
//  * 演示如何为 FP32 与 FP16 分别创建计算、布局描述。
//  * 程序在手写 4×4×4 小矩阵上运行，便于手动检验正确性。
//  环境要求：CUDA 11+，支持 Tensor Core 的 NVIDIA GPU。
// ------------------------------------------------------------

#include <hip/hip_runtime.h>
#include <hipblaslt.h>
#include <hip/hip_fp16.h>
#include <iostream>
#include <vector>
#include <iomanip>

// ------------------------------------------------------------
// 简易错误检查宏：捕获 CUDA Runtime 与 cuBLAS‑Lt 返回值
// ------------------------------------------------------------
#define CHECK_CUDA(call)                                                          \
    do {                                                                          \
        hipError_t status = (call);                                              \
        if (status != hipSuccess) {                                              \
            std::cerr << "CUDA error at line " << __LINE__ << ": "            \
                      << hipGetErrorString(status) << std::endl;                \
            exit(EXIT_FAILURE);                                                   \
        }                                                                         \
    } while (0)

#define CHECK_CUBLAS(call)                                                        \
    do {                                                                          \
        hipblasStatus_t status = (call);                                           \
        if (status != HIPBLAS_STATUS_SUCCESS) {                                    \
            std::cerr << "cuBLAS error at line " << __LINE__ << ": "          \
                      << status << std::endl;                                     \
            exit(EXIT_FAILURE);                                                   \
        }                                                                         \
    } while (0)

// ------------------------------------------------------------
// 朴素 CPU 矩阵乘：C = A * B
// 参数说明：
//   A: (M × K)  B: (K × N)  C: (M × N)
// ------------------------------------------------------------
void cpu_matmul(const float* A, const float* B, float* C, int M, int N, int K) {
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            float sum = 0.0f;
            for (int k = 0; k < K; ++k) {
                sum += A[i * K + k] * B[k * N + j];
            }
            C[i * N + j] = sum;
        }
    }
}

// ------------------------------------------------------------
// 矩阵打印辅助函数（按行主序遍历并打印）
// ------------------------------------------------------------
void print_matrix(const float* matrix, int rows, int cols, const char* name) {
    std::cout << "Matrix " << name << ":\n";
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            std::cout << std::setw(8) << std::fixed << std::setprecision(2)
                      << matrix[i * cols + j] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

int main() {
    //--------------------------------------------------------------------------
    // 1. 定义矩阵尺寸 (M × K) * (K × N) = (M × N)
    //--------------------------------------------------------------------------
    const int M = 4, K = 4, N = 4;

    //--------------------------------------------------------------------------
    // 2. 在主机端手动初始化矩阵 A、B
    //--------------------------------------------------------------------------
    float h_A[M * K] = {
        1.0f,  2.0f,  3.0f,  4.0f,
        5.0f,  6.0f,  7.0f,  8.0f,
        9.0f, 10.0f, 11.0f, 12.0f,
        13.0f,14.0f, 15.0f, 16.0f
    };

    // 刻意修改部分元素，确保 A ≠ B 以验证乘法
    float h_B[K * N] = {
        1.0f,  2.0f,  4.0f,  4.0f,  // 将 3.0f 改为 4.0f
        5.0f,  6.0f,  7.0f,  8.0f,
        9.0f, 10.0f, 11.0f, 12.0f,
        17.0f,18.0f, 19.0f, 20.0f   // 将最后一行改为 17~20
    };

    float h_C_cpu[M * N]      = {0}; // CPU 结果
    float h_C_gpu_fp32[M * N] = {0}; // GPU FP32 结果
    float h_C_gpu_fp16[M * N] = {0}; // GPU FP16（转回 FP32）结果

    // 打印输入矩阵，便于肉眼对比
    print_matrix(h_A, M, K, "A");
    print_matrix(h_B, K, N, "B");

    //--------------------------------------------------------------------------
    // 3. 在 GPU 上分配 FP32 / FP16 内存，并拷贝数据
    //--------------------------------------------------------------------------
    float *d_A_fp32, *d_B_fp32, *d_C_fp32;
    CHECK_CUDA(hipMalloc(&d_A_fp32, M * K * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_B_fp32, K * N * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_C_fp32, M * N * sizeof(float)));

    half *d_A_fp16, *d_B_fp16, *d_C_fp16;
    CHECK_CUDA(hipMalloc(&d_A_fp16, M * K * sizeof(half)));
    CHECK_CUDA(hipMalloc(&d_B_fp16, K * N * sizeof(half)));
    CHECK_CUDA(hipMalloc(&d_C_fp16, M * N * sizeof(half)));

    // 拷贝 FP32 数据到 GPU
    CHECK_CUDA(hipMemcpy(d_A_fp32, h_A, M * K * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B_fp32, h_B, K * N * sizeof(float), hipMemcpyHostToDevice));

    // 将 FP32 → FP16，并拷贝到 GPU
    std::vector<half> h_A_half(M * K), h_B_half(K * N);
    for (int i = 0; i < M * K; ++i) h_A_half[i] = __float2half(h_A[i]);
    for (int i = 0; i < K * N; ++i) h_B_half[i] = __float2half(h_B[i]);

    CHECK_CUDA(hipMemcpy(d_A_fp16, h_A_half.data(), M * K * sizeof(half), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B_fp16, h_B_half.data(), K * N * sizeof(half), hipMemcpyHostToDevice));

    //--------------------------------------------------------------------------
    // 4. 创建 cuBLAS‑Lt 句柄 & 矩阵/运算描述符
    //--------------------------------------------------------------------------
    hipblasLtHandle_t handle;
    CHECK_CUBLAS(hipblasLtCreate(&handle));

    // ---- 4.1  布局描述 (MatrixLayout) ----
    hipblasLtMatrixLayout_t matA_fp32, matB_fp32, matC_fp32;
    hipblasLtMatrixLayout_t matA_fp16, matB_fp16, matC_fp16;

    // 参数：数据类型、行数 (rows)、列数 (cols)、leading dim (ld)
    CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&matA_fp32, HIP_R_32F, K, M, K));
    CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&matB_fp32, HIP_R_32F, N, K, N));
    CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&matC_fp32, HIP_R_32F, N, M, N));

    CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&matA_fp16, HIP_R_16F, K, M, K));
    CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&matB_fp16, HIP_R_16F, N, K, N));
    CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&matC_fp16, HIP_R_16F, N, M, N));

    // ---- 4.2  乘法描述 (MatmulDesc) ----
    hipblasLtMatmulDesc_t matmulDesc_fp32, matmulDesc_fp16;
    CHECK_CUBLAS(hipblasLtMatmulDescCreate(&matmulDesc_fp32, HIPBLAS_COMPUTE_32F, HIP_R_32F));
    CHECK_CUBLAS(hipblasLtMatmulDescCreate(&matmulDesc_fp16, HIPBLAS_COMPUTE_16F, HIP_R_16F));

    // 设置 A、B 是否转置：此处均为常规 (No‑Transpose)
    hipblasOperation_t trans = HIPBLAS_OP_N;
    CHECK_CUBLAS(hipblasLtMatmulDescSetAttribute(matmulDesc_fp32, HIPBLASLT_MATMUL_DESC_TRANSA, &trans, sizeof(trans)));
    CHECK_CUBLAS(hipblasLtMatmulDescSetAttribute(matmulDesc_fp32, HIPBLASLT_MATMUL_DESC_TRANSB, &trans, sizeof(trans)));
    CHECK_CUBLAS(hipblasLtMatmulDescSetAttribute(matmulDesc_fp16, HIPBLASLT_MATMUL_DESC_TRANSA, &trans, sizeof(trans)));
    CHECK_CUBLAS(hipblasLtMatmulDescSetAttribute(matmulDesc_fp16, HIPBLASLT_MATMUL_DESC_TRANSB, &trans, sizeof(trans)));

    //--------------------------------------------------------------------------
    // 5. 调用 cublasLtMatmul 进行矩阵乘法
    //--------------------------------------------------------------------------
    const float alpha     = 1.0f;
    const float beta      = 0.0f;
    const half  alpha_h   = __float2half(1.0f);
    const half  beta_h    = __float2half(0.0f);

    // FP32 计算：C = alpha * B * A + beta * C
    CHECK_CUBLAS(hipblasLtMatmul(
        handle,
        matmulDesc_fp32,
        &alpha,
        d_B_fp32, matB_fp32,   // B 在左 (列主序 N × K)
        d_A_fp32, matA_fp32,   // A 在右 (列主序 K × M)
        &beta,
        d_C_fp32, matC_fp32,   // 输出 C (N × M)
        d_C_fp32, matC_fp32,   // 工作区复用输出
        nullptr, nullptr, 0, 0));

    // FP16 计算
    CHECK_CUBLAS(hipblasLtMatmul(
        handle,
        matmulDesc_fp16,
        &alpha_h,
        d_B_fp16, matB_fp16,
        d_A_fp16, matA_fp16,
        &beta_h,
        d_C_fp16, matC_fp16,
        d_C_fp16, matC_fp16,
        nullptr, nullptr, 0, 0));

    //--------------------------------------------------------------------------
    // 6. 将结果拷回主机，并把 FP16 → FP32 便于比较
    //--------------------------------------------------------------------------
    CHECK_CUDA(hipMemcpy(h_C_gpu_fp32, d_C_fp32, M * N * sizeof(float), hipMemcpyDeviceToHost));

    std::vector<half> h_C_gpu_fp16_half(M * N);
    CHECK_CUDA(hipMemcpy(h_C_gpu_fp16_half.data(), d_C_fp16, M * N * sizeof(half), hipMemcpyDeviceToHost));
    for (int i = 0; i < M * N; ++i) h_C_gpu_fp16[i] = __half2float(h_C_gpu_fp16_half[i]);

    //--------------------------------------------------------------------------
    // 7. CPU 参考结果
    //--------------------------------------------------------------------------
    cpu_matmul(h_A, h_B, h_C_cpu, M, N, K);

    //--------------------------------------------------------------------------
    // 8. 打印并验证
    //--------------------------------------------------------------------------
    print_matrix(h_C_cpu,      M, N, "C (CPU)");
    print_matrix(h_C_gpu_fp32, M, N, "C (GPU FP32)");
    print_matrix(h_C_gpu_fp16, M, N, "C (GPU FP16)");

    bool fp32_match = true, fp16_match = true;
    for (int i = 0; i < M * N; ++i) {
        if (std::abs(h_C_cpu[i] - h_C_gpu_fp32[i]) > 1e-5) fp32_match = false;
        if (std::abs(h_C_cpu[i] - h_C_gpu_fp16[i]) > 1e-2) fp16_match = false; // FP16 误差稍大
    }
    std::cout << "FP32 Results " << (fp32_match ? "match" : "do not match") << std::endl;
    std::cout << "FP16 Results " << (fp16_match ? "match" : "do not match") << std::endl;

    //--------------------------------------------------------------------------
    // 9. 资源释放
    //--------------------------------------------------------------------------
    CHECK_CUBLAS(hipblasLtMatrixLayoutDestroy(matA_fp32));
    CHECK_CUBLAS(hipblasLtMatrixLayoutDestroy(matB_fp32));
    CHECK_CUBLAS(hipblasLtMatrixLayoutDestroy(matC_fp32));
    CHECK_CUBLAS(hipblasLtMatrixLayoutDestroy(matA_fp16));
    CHECK_CUBLAS(hipblasLtMatrixLayoutDestroy(matB_fp16));
    CHECK_CUBLAS(hipblasLtMatrixLayoutDestroy(matC_fp16));
    CHECK_CUBLAS(hipblasLtMatmulDescDestroy(matmulDesc_fp32));
    CHECK_CUBLAS(hipblasLtMatmulDescDestroy(matmulDesc_fp16));
    CHECK_CUBLAS(hipblasLtDestroy(handle));
    CHECK_CUDA(hipFree(d_A_fp32));
    CHECK_CUDA(hipFree(d_B_fp32));
    CHECK_CUDA(hipFree(d_C_fp32));
    CHECK_CUDA(hipFree(d_A_fp16));
    CHECK_CUDA(hipFree(d_B_fp16));
    CHECK_CUDA(hipFree(d_C_fp16));

    return 0;
}
